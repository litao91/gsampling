#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

using namespace std;

float reduce_cpu(float* data, int* pat){
	
	float sum = 0;
	int m = pat[0];
	int numElement = pat[1];
		
	for(int i = 0; i < numElement; i++) 
	{
		float prod = 1;
		for(int j = 2; j < m+2; j++)
			prod *= data[pat[j]*numElement+i];
		sum += prod;
	}
	
	return sum;
}


__global__
void reduce_kernel1(float* d_out, float* d_in, const int size) {
	extern __shared__ float s_data[];
	const int tid = threadIdx.x;
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i >= size)
		s_data[tid] = 0;
	else	
		s_data[tid] = d_in[i];
	
	__syncthreads();

	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
	{
		if(tid < s) 
		{
			s_data[tid] += s_data[tid + s];
		}
		__syncthreads();
	}

	if(tid == 0) 
	{
		d_out[blockIdx.x] = s_data[0];
	}
}	

inline void swap(float* &p1, float* &p2) {

	float* tmp = p1;
	p1 = p2;
	p2 = tmp;

}


__global__
void reduce_kernel4(float* d_out, float* d_in, const int size) {

	extern __shared__ float s_data[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	s_data[tid] = 0;
	__syncthreads();
	if (i + blockDim.x < size )
		s_data[tid] = d_in[i] + d_in[i + blockDim.x];
	else if (i < size)
		s_data[tid] = d_in[i];
	__syncthreads();
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
		if(tid < s) {
			s_data[tid] += s_data[tid + s];
		}
		__syncthreads();
	}
	if(tid == 0) {d_out[blockIdx.x] = s_data[0];}
}	

float reduction(float* d_data, float* d_buf, const int numElement, int numThread, 
		void (*reduce_kernel)(float* d_out, float* d_data, const int size), 
		bool isKernel4 = false) {

	float* d_src = NULL;	//always store input data
	float* d_dst = NULL;	//always store output data

	//for the first level of reduction
	int n = numElement;
	d_src = d_data;
	d_dst = d_buf;

	int numBlock = numElement/numThread + (numElement%numThread? 1 : 0);
	int sharedMemSize = sizeof(float)*numThread;
	while(n > 1) {
		
		if(isKernel4) 
			numBlock = numBlock/2 + (numBlock%2 ? 1 : 0);
		reduce_kernel<<<numBlock, numThread, sharedMemSize>>>(d_dst, d_src, n);
		hipDeviceSynchronize();
		//for the next level		
		n = numBlock;
		numBlock = n/numThread+ (n%numThread ? 1 : 0);
		swap(d_dst, d_src);
	}

	hipDeviceSynchronize();
	swap(d_dst, d_src);
	float result = 0;
	hipMemcpy(&result, d_dst, sizeof(float), hipMemcpyDeviceToHost);
	return result;
}



__global__
void dot_kernel(float* dev_data, float* dev_dot, int* dev_pat) {
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	int m = dev_pat[0];
	int numElement = dev_pat[1];
	if (index < numElement)
	{
		float prod = 1;
		for(int j = 2; j < m+2; j++)
			prod = prod*dev_data[dev_pat[j]*numElement+index];
	
		dev_dot[index] = prod;
		__syncthreads();
	}
}


/*
*******************************************************************************************											
********************************************************************************************
*/

float* setdev_data(float* data, int dSize, int numElement)
{
	float* dev_data; //data��device�Ŀ���
	hipMalloc((void**)&dev_data, sizeof(float)*dSize);
	hipMemcpy(dev_data, data, sizeof(float)*dSize, hipMemcpyHostToDevice);
	return dev_data;
}

float* setdev_dot(int numElement)
{
	float* dev_dot;	//dev_dot����attribute֮���dot-product
	hipMalloc((void**)&dev_dot, sizeof(float)*numElement);
	return dev_dot;
}

float* setdev_out(int numElement)
{
	float* dev_out;
	hipMalloc((void**)&dev_out, sizeof(float)*numElement);
	return dev_out;
}

int* setdev_pat(int* pat, int m)
{
	int* dev_pat;//pat��device�Ŀ���
	hipMalloc((void**)& dev_pat, sizeof(int)*(m+2));
	hipMemcpy(dev_pat, pat, sizeof(int)*(m+2), hipMemcpyHostToDevice);
	return dev_pat;
}

float reduce_gpu(int numElement, int numThread, float* dev_data, float* dev_dot, float* dev_out, int* dev_pat)
{
	dot_kernel<<<numElement/numThread+(numElement%numThread ? 1 : 0),numThread>>>(dev_data, dev_dot, dev_pat);
	return reduction(dev_dot, dev_out, numElement, numThread, reduce_kernel4);	
}


void test_reduction() {

	////////////////////////////////////////////////////////////////////////�Ķ�/////////////////////////////////////////////

	const int numElement = 512*500;
	const int numAttribute = 100;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	//const int numElement = 1024*1024;
	//const int numAttribute = 100;
	const int dSize = numElement*numAttribute;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	////////////////////////////////////////////////////////////////////////�Ķ�/////////////////////////////////////////////
	//Data Generator for the Table
	float* data = (float*)malloc(sizeof(float)*dSize);
	for(int i = 0; i < dSize; i++) 
	{
		//data[i] = (float)rand()/RAND_MAX;
		data[i] = 0.5;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	const int numThread = 512;
	float elapsedTime = 0.0f;
	float result = 0;
	float* dev_data; //data��device�Ŀ���
	hipMalloc((void**)&dev_data, sizeof(float)*dSize);
	hipMemcpy(dev_data, data, sizeof(float)*dSize, hipMemcpyHostToDevice);
	float* dev_dot;	//dev_dot����attribute֮���dot-product
	hipMalloc((void**)&dev_dot, sizeof(float)*numElement);
	float* dev_out;
	hipMalloc((void**)&dev_out, sizeof(float)*numElement);

	////////////////////////////////////////////////////////////////////////�Ķ�/////////////////////////////////////////////
	int m = 3; //��3��attribute��pattern
	int* pat = (int *)malloc(sizeof(int)*(m+2));
	pat[0] = m; pat[1] = numElement; pat[2] = 1; pat[3] = 2; pat[4] = 3;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	int* dev_pat;//pat��device�Ŀ���
	hipMalloc((void**)& dev_pat, sizeof(int)*(m+2));
	hipMemcpy(dev_pat, pat, sizeof(int)*(m+2), hipMemcpyHostToDevice);
	
	//**************************************CPU*******************************************
	hipEventRecord(start, 0);
	printf("numElement = %d\n", numElement);
	printf("reduce_cpu result: %f\n", reduce_cpu(data, pat));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("cpu elapsed time: %.3f ms\n", elapsedTime);
	printf("---------------------------------\n\n");
	
	//************************************GPU******************************************
	elapsedTime = 0.0f;	
	hipEventRecord(start, 0);
	dot_kernel<<<numElement/numThread+(numElement%numThread ? 1 : 0),numThread>>>(dev_data, dev_dot, dev_pat);
	
//	float* dot = (float*)malloc(sizeof(float)*numElement);
//	hipMemcpy(dot, dev_dot, sizeof(float)*numElement, hipMemcpyDeviceToHost);
//	float testsum = 0;
//	for (int i = 0; i < numElement; i++)
//		testsum += dot[i];
		
//	cout<<"dot result = "<<testsum<<endl;
	
	result = reduction(dev_dot, dev_out, numElement, numThread, reduce_kernel4);
	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel1 elapsed time: %.3f ms\n", elapsedTime);
	printf("kernel1 result: %f\n\n", result);

	hipFree(dev_data);
	hipFree(dev_out);
	hipFree(dev_dot);
	free(data);

}


